#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <chrono>
using namespace std;

#define HANDLE_ERROR(error) { \
    if (error != hipSuccess) { \
        fprintf(stderr, "%s in %s at line %d\n", \
                hipGetErrorString(error), __FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}


const int matrixSize = 1000;
const int threads = 51;
int* m1, *m2, *resultAll;


void init()
{
	m1 = new int[matrixSize * matrixSize];
	m2 = new int[matrixSize * matrixSize];
	resultAll = new int[matrixSize * matrixSize];
	for (int i = 0; i < matrixSize; i++)
	{
		for (int j = 0; j < matrixSize; ++j)
		{
			m1[i* matrixSize + j] = rand() % 100;
			m2[i* matrixSize + j] = rand() % 100;
			resultAll[i* matrixSize + j] = 0;
		}
	}
}

void nullResult()
{
	for (int i = 0; i < matrixSize; i++)
	{
		for (int j = 0; j < matrixSize; ++j) {
			resultAll[i* matrixSize + j] = 0;
		}
	}
}

void print(int* matrix)
{

	for (int i = 0; i < matrixSize; i++)
	{
		for (int j = 0; j < matrixSize; j++)
		{

			cout << matrix[i* matrixSize + j] << "\t";

		}
		cout << endl;
	}
}

__global__ void multiplyKernel(int *c, const int *a, const int *b)
{
	
	for (int i = threadIdx.x; i < matrixSize; i += threads) 
	{
		for (int j = 0; j < matrixSize; j++)
		{
			for (int k = 0; k < matrixSize; k++)
			{
				c[i*matrixSize + j] += a[i*matrixSize + k] * b[k*matrixSize + j];
			}
		}
	}
}

void multiplyLinear(int *c, const int *a, const int *b)
{
	for (int i = 0; i < matrixSize; i++)
	{
		for (int j = 0; j < matrixSize; j++)
		{
			for (int k = 0; k < matrixSize; k++)
			{
				c[i*matrixSize + j] += a[i*matrixSize + k] * b[k*matrixSize + j];
			}
		}
	}
}

void MatrixMultWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;

	HANDLE_ERROR(hipSetDevice(0));

	HANDLE_ERROR(hipMalloc((void**)&dev_c, size * size * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_a, size * size * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, size * size * sizeof(int)));

	HANDLE_ERROR(hipMemcpy(dev_a, a, size * size * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, size * size * sizeof(int), hipMemcpyHostToDevice));

	multiplyKernel << <1, threads >> > (dev_c, dev_a, dev_b);

	HANDLE_ERROR(hipGetLastError());
	HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR(hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_c));
}

std::chrono::high_resolution_clock::time_point currentTime()
{
	return  std::chrono::high_resolution_clock::now();
}

float timeElapsed(std::chrono::high_resolution_clock::time_point startTime)
{
	auto endTime = currentTime();
	auto resT = chrono::duration_cast<chrono::milliseconds>(endTime - startTime);
	return resT.count();
}

void main()
{
	srand((unsigned)time(NULL));

	init();
	//print(m1);
	//cout << endl << endl;

	//print(m2);
	//cout << endl;

	auto startTime = currentTime();
	//multiplyLinear(resultAll, m1, m2);
	auto Linear = timeElapsed(startTime);
	cout << "Linear time: \t" << Linear << endl << endl;

	//print(resultAll);

	nullResult();

	startTime = currentTime();
	MatrixMultWithCuda(resultAll, m1, m2, matrixSize);
	auto Threaded = timeElapsed(startTime);
	cout << "Threaded time: \t" << Threaded << endl << endl;

	//print(resultAll);

	cout << "Acceleration: " << Linear / Threaded << endl;
	cout << "Efficiency: " << Linear / (Threaded*matrixSize) << endl;


	HANDLE_ERROR(hipDeviceReset());
}



